#include "hip/hip_runtime.h"
#include <algorithm>

#include "caffe/common.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

template <typename Dtype>
__global__ void reduce_precision_gpu_kernel(Dtype* data, size_t size, const unsigned int prec, const float scale, const int quantizer) {
  // quantizers 
  // 0: mid rise using all signed integers
  // 1: mid tread using all signed integers
  // 2: mid tread with a symmetric range (odd number of bins)
  // 3: 2 with a squished zero bin for only 0 values
  int midrise   = quantizer < 1;
  int odd       = quantizer > 1;
  int zero_bin  = quantizer > 2;
  int intmax;
  int intmin;

  intmax = (1 << (prec-1)) - 1;
  if (odd){
    intmin = -1 * intmax; // -1, 0, 1
  } else {
    intmin = - (1 << (prec-1)); // -2, -1, 0, 1
  }

  CUDA_KERNEL_LOOP(index, size) {
    Dtype d = data[index];
    int q;

    // move near zero values to first non zero bins
    if (zero_bin){
      if (d > 0 && d < scale) // is this right?
        d = scale;
      else if (d < 0 && d > -scale)
        d = -scale;
    }

    // quantize
    if (midrise) {
      q = floor(d * scale);
    } else {
      q = floor(d * scale + 0.5);
    }
    // clamp
    if (q > intmax) q = intmax;
    if (q < intmin) q = intmin;

    // reconstruct
    if (d == 0 && zero_bin) {
      d = 0;
    } else if (midrise) {
      d = ( q + 0.5 ) / scale;
    } else {
      d = ( q ) / scale;
    }

    data[index] = d;
  }
}

template <typename Dtype>
void reduce_precision_gpu(Dtype* data, size_t size, const unsigned int prec, const float scale, const int quantizer) {
  size_t num_kernels = size;
  reduce_precision_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(data, size, prec, scale, quantizer);
}

template void reduce_precision_gpu<float>(float* data, size_t size, const unsigned int prec, const float scale, const int quantizer);
template void reduce_precision_gpu<double>(double* data, size_t size, const unsigned int prec, const float scale, const int quantizer);


}  // namespace caffe
